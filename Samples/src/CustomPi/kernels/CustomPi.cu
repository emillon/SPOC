
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif
struct point{
	float x;
	float y;
};

__global__ void pi(const struct point* A, float* res, const int nbPoint, const float ray){
	  const int idx = 32*blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < nbPoint-32*blockDim.x)
	{//blockDim.x * blockIdx.x + threadIdx.x;
	   const int i1 =  idx+blockDim.x;
	   const int i2 =  i1+blockDim.x;
	   const int i3 =  i2+blockDim.x;
	   const int i4 =  i3+blockDim.x;
	   const int i5 =  i4+blockDim.x;
	   const int i6 =  i5+blockDim.x;
	   const int i7 =  i6+blockDim.x;
	   const int i8 =  i7+blockDim.x;
	   const int i9 =  i8+blockDim.x;
	   const int i10 =  i9+blockDim.x;
	   const int i11 =  i10+blockDim.x;
	   const int i12 =  i11+blockDim.x;
	   const int i13 =  i12+blockDim.x;
	   const int i14 =  i13+blockDim.x;
	   const int i15 =  i14+blockDim.x;
	   const int i16 =  i15+blockDim.x;
	   const int i17 =  i16+blockDim.x;
	   const int i18 =  i17+blockDim.x;
	   const int i19 =  i18+blockDim.x;
	   const int i20 =  i19+blockDim.x;
	   const int i21 =  i20+blockDim.x;
	   const int i22 =  i21+blockDim.x;
	   const int i23 =  i22+blockDim.x;
	   const int i24 =  i23+blockDim.x;
	   const int i25 =  i24+blockDim.x;
	   const int i26 =  i25+blockDim.x;
	   const int i27 =  i26+blockDim.x;
	   const int i28 =  i27+blockDim.x;
	   const int i29 =  i28+blockDim.x;
	   const int i30 =  i29+blockDim.x;
	   const int i31 =  i30+blockDim.x;
	//int i =  idx*blockDim.x;
	res[idx] =  (A[idx].x*A[idx].x + A[idx].y*A[idx].y <= ray);
	res[i1] =  (A[i1].x*A[i1].x + A[i1].y*A[i1].y <= ray);
	res[i2] =  (A[i2].x*A[i2].x + A[i2].y*A[i2].y <= ray);
	res[i3] =  (A[i3].x*A[i3].x + A[i3].y*A[i3].y <= ray);
	res[i4] =  (A[i4].x*A[i4].x + A[i4].y*A[i4].y <= ray);
	res[i5] =  (A[i5].x*A[i5].x + A[i5].y*A[i5].y <= ray);
	res[i6] =  (A[i6].x*A[i6].x + A[i6].y*A[i6].y <= ray);
	res[i7] =  (A[i7].x*A[i7].x + A[i7].y*A[i7].y <= ray);

	res[i8] =  (A[i8].x*A[i8].x + A[i8].y*A[i8].y <= ray);
	res[i9] =  (A[i9].x*A[i9].x + A[i9].y*A[i9].y <= ray);
	res[i10] =  (A[i10].x*A[i10].x + A[i10].y*A[i10].y <= ray);
	res[i11] =  (A[i11].x*A[i11].x + A[i11].y*A[i11].y <= ray);
	res[i12] =  (A[i12].x*A[i12].x + A[i12].y*A[i12].y <= ray);
	res[i13] =  (A[i13].x*A[i13].x + A[i13].y*A[i13].y <= ray);
	res[i14] =  (A[i14].x*A[i14].x + A[i14].y*A[i14].y <= ray);
	res[i15] =  (A[i15].x*A[i15].x + A[i15].y*A[i15].y <= ray);

	res[i16] =  (A[i16].x*A[i16].x + A[i16].y*A[i16].y <= ray);
	res[i17] =  (A[i17].x*A[i17].x + A[i17].y*A[i17].y <= ray);
	res[i18] =  (A[i18].x*A[i18].x + A[i18].y*A[i18].y <= ray);
	res[i19] =  (A[i19].x*A[i19].x + A[i19].y*A[i19].y <= ray);
	res[i20] =  (A[i20].x*A[i20].x + A[i20].y*A[i20].y <= ray);
	res[i21] =  (A[i21].x*A[i21].x + A[i21].y*A[i21].y <= ray);
	res[i22] =  (A[i22].x*A[i22].x + A[i22].y*A[i22].y <= ray);
	res[i23] =  (A[i23].x*A[i23].x + A[i23].y*A[i23].y <= ray);

	res[i24] =  (A[i24].x*A[i24].x + A[i24].y*A[i24].y <= ray);
	res[i25] =  (A[i25].x*A[i25].x + A[i25].y*A[i25].y <= ray);
	res[i26] =  (A[i26].x*A[i26].x + A[i26].y*A[i26].y <= ray);
	res[i27] =  (A[i27].x*A[i27].x + A[i27].y*A[i27].y <= ray);
	res[i28] =  (A[i28].x*A[i28].x + A[i28].y*A[i28].y <= ray);
	res[i29] =  (A[i29].x*A[i29].x + A[i29].y*A[i29].y <= ray);
	res[i30] =  (A[i30].x*A[i30].x + A[i30].y*A[i30].y <= ray);
	res[i31] =  (A[i31].x*A[i31].x + A[i31].y*A[i31].y <= ray);
	}
}

#ifdef __cplusplus
}
#endif
